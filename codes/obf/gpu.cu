
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ unsigned A(unsigned a, unsigned b)
{
    while( a & b )
    {
        unsigned X = a ^ b;
        unsigned Y = (a&b)<<1;

        a = X;
        b = Y;
    }
    return a ^ b;
}

__device__ unsigned G(unsigned a, unsigned b)
{
    for(;b;b^=a^=b^=a%=b);
    return !--a;
}

__device__ unsigned F(unsigned a, unsigned b)
{
    unsigned s = 0;
    while( b )
    {
        s += G(a, b);
        b--;
    }
    return s;
}

__device__ unsigned S(unsigned a, unsigned b)
{
    for( ; ; )
    {
        if ( a )
        {
            if ( a >= b )
            {
                unsigned x = A(a, 1+~b);
                unsigned y = b + 2;

                a = x;
                b = y;
            }
            else return 0;
        }
        else return 1;
    }
}

__global__ void kernel(char* result)
{
    int t, sum, x = blockIdx.x*blockDim.x + threadIdx.x + 1;
    int X = x;
    for( sum=0; X; X/=10 ) sum += X % 10;
    if ( x % sum ) t = 0;
    else if ( S(F(x,x),1) ) t = 1;
    else t = 2;
    result[x] = t;
}

int main()
{
    char str[10][10] = { ":Ugly", ":Good", ":Bad"};
    char *devR;
    char result[200005];

    hipMalloc( (void**)&devR, 200005 );

    kernel<<<200, 1000>>>(devR);

    hipMemcpy( result, devR, 200005, hipMemcpyDeviceToHost );

    for( unsigned x=1; x<=200000; x++ )
    {
        printf("%d", x);
        puts( str[result[x]] );
    }

    printf("Who's 25?\n");

    hipFree( devR );
    return 0;
}
